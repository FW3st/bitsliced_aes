#include "hip/hip_runtime.h"
#define TEST
#include "bits_aes.cu"
#include "aes_cbc.h"

void printError(){
    hipError_t error = hipGetLastError ();
    printf("error: %s\n",hipGetErrorName(error) );
}

void fill_random(int* ar, int len){
    for(int i=0; i<len; i++){
        ar[i] = rand();
    }
}

__global__ void __test_bitorder_transform(char*plain, uint128_t transformed[8]){
    bitorder_transform(plain, transformed);
}

void check_bitorder(char (*raw)[16], char (*ord)[16]){
    char x,y;
    for(int by=0; by<16; by++){
        for(int bi=0; bi<8; bi++){
            for(int d=0; d<8; d++){
                x = (raw[d][by]>>bi)&1;
                y = (ord[bi][by]>>(7-d))&1;
                if(x!=y){
                    printf("check_bitorder failed\n");
                    printf("x:%i, y:%i\n", x, y);
                    printf("by:%i, by:%i, d:%i\n", by, bi,d);
                    return;
                }
            }
        }
    }
    printf("check_bitorder passed\n");
}

__global__ void __test_bitorder_retransform(char*plain, uint128_t transformed[8]){
     bitorder_retransform(plain, transformed);
}

void check_bitreorder(char* raw, char* reo){
    if(memcmp(raw, reo, 16*8) != 0){
        printf("check_bitreorder failed\n");
        return;
    }
    printf("check_bitreorder passed\n");
}

uint128_t touint128(void* ar){
    uint64_t* inp = (uint64_t*) ar;
    uint128_t ret;
    ret.lo = inp[0];
    ret.hi = inp[1];
    return ret;
}


void check_encrypt(char* plain, char* key){
    //intel avx cbc aes, n times without IV ~> ecb
    struct cbc_key_data round_keys;
    uint8_t iv[16];
    uint8_t out[16*8];
    memset(iv, 0, 128);
    aes_cbc_precomp((uint8_t*)key,CBC_128_BITS,&round_keys);
    for(int i=0; i<8; i++){
        aes_cbc_enc_128(plain+16*i, iv, round_keys.enc_keys,out+16*i, 16);
    }
    
    //bitsliced aes
    uint8_t* bs_out[16*8];
    char *d_plain;
    uint128_t d_roundkey[11][8];
    char *d_cypher;
    char* bs_roundkey = (char*) malloc(1408);
    create_round_key(key, bs_roundkey);
    
    hipMalloc((void**)&d_plain, 16*8);
    hipMalloc((void**)&d_cypher, 16*8);
    hipMalloc((void**)&d_roundkey, 1408);
    
    hipMemcpy(d_plain, plain, 16*8, hipMemcpyHostToDevice);
    hipMemcpy(d_roundkey, bs_roundkey, 1408, hipMemcpyHostToDevice);
    
    encrypt<<<NUM_BLOCKS,1>>>(d_plain, d_roundkey, d_cypher);
    
    hipMemcpy(bs_out, d_cypher, 16*8, hipMemcpyDeviceToHost);
    
    if(memcpy(bs_out,out,16*8)==0){
        printf("Encrypt passed\n");
    } else {
        printf("Encrypt failed\n");
    }
}

int main(void) {
    time_t t;
    srand((unsigned) time(&t));
    uint128_t inp128[8];
    uint128_t inp1282[8];
    uint128_t out128[8];
    uint128_t out1282[8];
    uint128_t* inp128_cuda;
    uint128_t* out128_cuda;
    uint128_t* out128_cuda2;
    
    char (*raw)[16] = (char(*)[16]) malloc(16*8);
    char (*raw2)[16] = (char(*)[16]) malloc(16*8);
    int* ran_buf = (int*) raw;
    int* ran_buf2 = (int*) raw2;
    //memset(ran_buf,0,16*8);
    //raw[0][0] = 0x80;
    for(int i = 0; i<8; i++){
        fill_random(ran_buf, 32);
        fill_random(ran_buf2, 32);
        inp128[i] = touint128(ran_buf+4*i);
        inp1282[i] = touint128(ran_buf2+4*i);
    }        
    
    hipMalloc((void**)&inp128_cuda, 16*8);
    hipMalloc((void**)&out128_cuda, 16*8);
    hipMalloc((void**)&out128_cuda2, 16*8);

    // CHECK bitorder_transform
    hipMemcpy(inp128_cuda, inp128, 16*8, hipMemcpyHostToDevice);
    __test_bitorder_transform<<<1,1>>>((char*)((void*)inp128_cuda),out128_cuda);
    hipMemcpy(out128, out128_cuda, 16*8, hipMemcpyDeviceToHost);
    check_bitorder( (char(*)[16]) ((void*)inp128), (char(*)[16]) ((void*)out128));
    
    // CHECK bitorder_retransform
    __test_bitorder_retransform<<<1,1>>>((char*)((void*)out128_cuda2),out128_cuda);
    hipMemcpy(out1282, out128_cuda2, 16*8, hipMemcpyDeviceToHost);
    check_bitreorder((char(*)) ((void*)inp128), (char(*)) ((void*)out1282));
    
    
    // CHECK encrypt
    check_encrypt((char*)raw, (char*)raw2);
    //hipDeviceSynchronize();
    
    hipFree(out128_cuda2);
    hipFree(out128_cuda);
    hipFree(inp128_cuda);
    return 0;
}